#include "hip/hip_runtime.h"
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm> 
#include <cstdlib>  
#include <hip/hip_runtime.h>

//used for norm function
//#include <hipblas.h>
#include <hipblas.h>

//used for conversion from coo to csr sparse matrix formats
#include <hipsparse.h>

#define BLOCKS 512
#define THREADS 1024
#define THREAD_INC 32


__global__ void kernel_solve_v0(double *val, int *row, int *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	*sum = 0.0;
	
	for (int ir = 0; ir < n; ir++) {
		
		if (row[ir] == ix) {
			if (col[ir] == ix) {
				*aii = val[ir];
				//printf("ix is: %d   col: %d  \n", ix, col[ir]);
			}
			else {
				*sum += (val[ir] * x0[col[ir]]); //sum non-zero values in row vec
				//printf("sum is: %f  ", *sum);
			}
		}
	}
}

__global__ void kernel_solve_v1(double *val, int *row, int *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	extern __shared__ double thread_sum[]; //access dynamic shared mem


	//initialize thread_sum;
	for (int ir = threadIdx.x; ir < n; ir = ir + blockDim.x) { thread_sum[ir] = 0; }

	__syncthreads();


	for (int ir = threadIdx.x; ir < n; ir = ir + blockDim.x) {
		if (row[ir] == ix) {

			if (col[ir] == ix) {
				*aii = val[ir];
			}
			else
			{
				thread_sum[ir] = ((val[ir]) * x0[col[ir]]); //sum non-zero values in row vec
				//printf("this is row %0.f at thread %d: val at index is %f\n", row[ir], ir, val[ir]);
			}
		}
	}

	__syncthreads();

	//sum all threads and save them to the sum pointer
	if (threadIdx.x == 0) {
		double total = 0.0;
		for (int i = 0; i < n; i++) {
			total += thread_sum[i];
		}
		*sum = total; //return result
	}
}

__global__ void kernel_solve_v2(double *val, int *row, int *col, double* b, double* x0, double* x, double* xdiff, unsigned int n, unsigned int m, hipblasHandle_t blas_handle, bool *end_flag) {
	double sum = 0.0;
	double aii = 0.0;
	int indx = 0, next = 0;

	//for (int ix = threadIdx.x; ix < m; ix = ix + blockDim.x) {
	for (int ix = 0; ix < m; ix++) {

			indx = row[ix];
			next = row[ix + 1];
			//printf("%d:%d      ", ix, indx);
			//printf("at x %d is %d\n", ix, col[indx]);
			sum = 0.0;
			while (indx < next) {
				if (col[indx] == ix) { //condition is never met
					aii = val[indx];
					//printf("here sum shuld be 0 %d\n", sum);
				}
				else
				{
					sum += ((val[indx]) * x0[col[indx]]); //sum non-zero values in row vec
														  //printf("AT x %d, sum is %d\n", ix, sum);
				}
				indx++;
			}

			x[ix] = (b[ix] - sum) / aii;
			xdiff[ix] = x[ix] - x0[ix];
			printf("Jacobi iteration %d - error is: %f\r", ix, xdiff[ix]); //ADD THIS BACK IN
			//if (ix < 10) {
			//	printf("row: %d, x is: %f, sum is: %d\n", ix, x[ix], sum);
			//}
		} //end for (vals of x)

	//if (threadIdx.x == 0) {
	//	
	//	//compute the norm of vec x
		//hipblasDnrm2(blas_handle, m, x, m, x_norm);
	//	//compute the norm of vec x_diff
	//	hipblasDnrm2(blas_handle, m, xdiff, m, x_diffnorm);

	//	if (*x_diffnorm / *x_norm < 1e-8) {
	//		*end_flag = true;
	//		printf("CONVERGENCE");
	//		return;
	//	}
	//}
	
	
}

__global__ void init_coo(long long *lrow, long long *lcol, int* row, int* col, const unsigned int n) {
	for (int i = threadIdx.x; i < n; i = i + blockDim.x) {

		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
		//if (i < 20) {
		//	printf("%d, ", row[i]);
		//}
	}
}


int* convert_csr(hipsparseHandle_t sparse_handle, const int *cooRowInd, int *csrRowPtr, int n, int m) {
	
	int *d_rowPtr;

	//hipMalloc((void **)&d_rowPtr, sizeof(int) * m+1);
	//hipMemcpy(d_rowPtr, csrRowPtr, sizeof(int) * m+1, hipMemcpyHostToDevice);


	//for (int i = 0; i < 10; i++) {
	//	printf("%d\n", cooRowInd[i]);
	//	//csrRowPtr[i] = 1;
	//}
	hipsparseStatus_t s;
	s = hipsparseXcoo2csr(sparse_handle, cooRowInd, n, m, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(csrRowPtr, d_rowPtr, sizeof(double) * m+1, hipMemcpyDeviceToHost);
	//printf("%d\n", *csrRowPtr);
	//if (s == HIPSPARSE_STATUS_SUCCESS) {
	//	for (int i = 0; i < 20; i++) {
	//		printf("%d, ", csrRowPtr[i]);
	//	}
	//}
	return csrRowPtr;
}

int determine_threads(int n, int m) {
	int threads = n / THREAD_INC;
	if ((threads*THREAD_INC) + THREAD_INC <= 1024) {
		threads = (threads*THREAD_INC) + THREAD_INC;
	}
	else {
		threads = 1024;
	}

	if (n * sizeof(double) > 48 * 1024) {
		printf("IMAGE TOO LARGE > EXITING");
		exit(EXIT_FAILURE);
	}

	return threads;
}

double compute_error(double* x, double* x_diff, double* d_x, double* d_xdiff, int x_size, hipblasHandle_t blas_handle) {
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	hipMemcpy(d_x, x, (sizeof(double) * x_size), hipMemcpyHostToDevice);
	hipMemcpy(d_xdiff, x_diff, (sizeof(double) * x_size), hipMemcpyHostToDevice);

	//compute the norm of vec x
	hipblasDnrm2(blas_handle, x_size, d_x, x_size, x_norm);
	//compute the norm of vec x_diff
	hipblasDnrm2(blas_handle, x_size, d_xdiff, x_size, x_diffnorm);

	return *x_diffnorm / *x_norm;
}

double* parallel_solve_v2(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter, double tol) {
	//for norm function
	hipblasHandle_t blas_handle;
	hipsparseHandle_t sparse_handle;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&sparse_handle);
	hipblasCreate(&blas_handle);
	hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//allocate memory of local variables
	bool *end_flag = (bool*)calloc(1, sizeof(bool));
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	//double* sum = (double*)calloc(1, sizeof(double));
	//double* aii = (double*)calloc(1, sizeof(double));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//declare pointers to device memory
	bool *d_flag;
	long long *d_lrow, *d_lcol;
	double *d_val, *d_x0, *d_b, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_flag, sizeof(double));
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_lrow, sizeof(long long) * n);
	hipMalloc((void **)&d_lcol, sizeof(long long) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	//hipMalloc((void **)&d_sum, sizeof(double));
	//hipMalloc((void **)&d_aii, sizeof(double));
	hipMalloc((void **)&d_b, sizeof(double) * x_size);
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lcol, lcol, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lrow, lrow, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, (sizeof(double) * x_size), hipMemcpyHostToDevice);
	//hipMemcpy(d_x, x, (sizeof(double) * n), hipMemcpyHostToDevice);
	//hipMemcpy(d_xdiff, x_diff, (sizeof(double) * n), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	//printf("Size of X: %d\n", x_size);
	int threads = determine_threads(x_size, x_size);
	//int threads = x_size; //what do you think Jacob?
	printf("Solving using parallel Jacobi method\nCurrent matrix contains %d coefficients, solving for %d values\n%d threads assigned\n", n, x_size, threads);
	//printf("threads: %d\n", threads);

	//convert input row & col vectors into integer vectors
	init_coo << <1, n >> > (d_lrow, d_lcol, d_row, d_col, n);


	//this is just for testing
	//hipMemcpy(row, d_row, (sizeof(int) * n), hipMemcpyDeviceToHost);
	//hipMemcpy(col, d_col, (sizeof(int) * n), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();


	//int *csrRowPtr = (int*)calloc(x_size + 1, sizeof(int));
	int *d_rowPtr = 0;

	//for (int i = 0; i < (x_size + 1); i++) {
	//	printf(":%d %d\n", row[i],i);
	//}

	hipMalloc((void**)&d_rowPtr, (x_size + 1) * sizeof(int));
	hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, d_rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(row, d_rowPtr, (sizeof(int) * (x_size + 1)), hipMemcpyDeviceToHost);

	//for (int i = 0; i < (x_size + 1); i++) {
	//	printf(":%d\n", row[i]);
	//}

	//row = convert_csr(sparse_handle, row, n, x_size);
	hipFree(d_lrow);
	hipFree(d_lcol);
	hipFree(d_rowPtr);

	for (int i = 0; i < max_iter; i++) {
		*end_flag = false;
		//printf("x at iter %d is %f : %f\n", i, *x, *x0);

		std::swap(x0, x);
		hipMemcpy(d_x0, x0, (sizeof(double) * x_size), hipMemcpyHostToDevice);
		
		kernel_solve_v2 << <1, threads>> > (d_val, d_row, d_col, d_b, d_x0, d_x, d_xdiff, n, x_size, blas_handle, end_flag);

		hipMemcpy(x, d_x, sizeof(double) * x_size, hipMemcpyDeviceToHost);
		hipMemcpy(end_flag, d_flag, sizeof(bool), hipMemcpyDeviceToHost);

		if (*end_flag) { break; };
	} //end for (iterations)
	printf("\n");

	  //wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);
	//hipFree(d_sum);
	//hipFree(d_aii);

	return x;
}

double* parallel_solve_v1(double *val, long long *lrow, long long *lcol, double *b, int *fail, const unsigned int n, const unsigned int x_size, const unsigned int max_iter, double tol) {
	//for norm function
	hipblasHandle_t blas_handle;
	hipblasCreate(&blas_handle);


	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//allocate memory of local variables
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	double* sum = (double*)calloc(1, sizeof(double));
	double* aii = (double*)calloc(1, sizeof(double));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));

	//declare pointers to device memory
	double *d_val, *d_x0, *d_sum, *d_aii, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	hipMalloc((void **)&d_sum, sizeof(double));
	hipMalloc((void **)&d_aii, sizeof(double));
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);


	for (int i = 0; i < n; i++) {
		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
		//printf("col: %d\n", row[i]);
	}

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_col, col, (sizeof(int) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_row, row, (sizeof(int) * n), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	int threads = determine_threads(n, x_size);
	printf("Solving using parallel Jacobi method\nCurrent matrix contains %d coefficients, solving for %d values\n%d threads assigned\n", n, x_size, threads);

	//convert input row & col vectors into integer vectors
	//init_coo << <1, threads >> > ( d_lrow, d_lcol, d_row, d_col, n);
	

	//hipMemcpy(row, d_row, (sizeof(int) * n), hipMemcpyDeviceToHost);
	//hipMemcpy(col, d_col, (sizeof(int) * n), hipMemcpyDeviceToHost);

	////hipDeviceSynchronize();

	//for (int ib = 0; ib < 20; ib++) {
	//	printf("%d, ", row[ib]);
	//}

	//int *csrRowPtr = 0;

	//csrRowPtr = (int*)calloc(x_size + 1, sizeof(int));

	//
	//hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, d_rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	//hipMemcpy(csrRowPtr, d_rowPtr, (sizeof(int) * (x_size + 1)), hipMemcpyDeviceToHost);


	
	//hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	//csrRowPtr = convert_csr(sparse_handle, row, csrRowPtr, n, x_size);

	//for (int ib = 0; ib < 20; ib++) {
	//	printf("(%d, ", col[ib]);
	//	printf("%d)", row[ib]);
	//}
	//hipFree(d_lrow);
	//hipFree(d_lcol);


	/* Here is full program that is set for parallel running. It always produces sum = 0. There may be an issue with copying 
	mem from device in a loop. We may need to re-copy back to device for each iteration. currently testing with full-fat but
	I need a testing strategy to break it up and check the individual components.*/

	/*At the 3rd iteration, (x=2) the graphics crashes and the outputs start being wrong (I think the actual kernel stops
	executing and the loop continues printing and updating x[ix]*/

	/*first 2 iterations of x are correct. From there x goes off some threshold. This is a memory problem. I need some way to
	free up x0 memory when I'm done each iteration because its eating up all the space.*/

	/*Results vary from totally correct to compeletely random with every run. I have no idea what is causing this.*/

	/*Problem was the order that the memcpy happened*/
	
	for (int i = 0; i < max_iter; i++) {
		//printf("x at iter %d is %f : %f\n", i, *x, *x0);

		//x0 = x;
		std::swap(x0, x);
		hipMemcpy(d_x0, x0, (sizeof(double) * x_size), hipMemcpyHostToDevice);

		for (int ix = 0; ix < x_size; ix++) {
			*sum = 0.0;

				//execute solve in parallel and allocate n doubles of shared memory
				kernel_solve_v1 << <1, threads, n * sizeof(double) >> > (d_val, d_row, d_col, d_x0, d_sum, d_aii, ix, n);

				//copy shared memory to host (OUTPUTS)
				hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(aii, d_aii, sizeof(double), hipMemcpyDeviceToHost);
				
			x[ix] = (b[ix] - *sum) / *aii;
			x_diff[ix] = x[ix] - x0[ix];
			if (ix < 10) {
				//printf("row: %d, x is: %f:%f, sum is: %f\n", ix, x[ix], x0[ix], *sum);
			}
		} //end for (vals of x)

		//calculate error between x and x_diff
		double error = compute_error(x, x_diff, d_x, d_xdiff, x_size, blas_handle);
		printf("Jacobi iteration %d - error is: %f\r", i, error); //ADD THIS BACK IN
		////printf("iteration: %d error: %f\n", i, error);
		if (error <= tol) {
			*fail = 0;
			printf("Convergence reached at iter %d, val %f\n", i, *x);
			break; //convergence reached
		}
	} //end for (iterations)

	//wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);
	hipFree(d_sum);
	hipFree(d_aii);

	return x;
}

double* jacobi_serial(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	//allocate output vectors x and x0
	double *x = new double[x_size];
	double *x0 = new double[x_size];
	double sum = 0.0, aii = 0.0;

	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));

	for (int i = 0; i < n; i++) {
		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
		//printf("%d\n", row[i]);
	}

	for (int i = 0; i < max_iter; i++) {
		x0 = x;
		//printf("x at iter %d is %f\n", i, *x);

		for (int ix = 0; ix < x_size; ix++) {
			sum = 0; //reset sum
			
			for (int ir = 0; ir < n; ir++) {
				if (row[ir] == ix) {
					if (col[ir] == ix) {
						aii = val[ir];
					}
					else {
						sum += (val[ir] * x0[col[ir]]); //sum non-zero values in row vec
					}
				}
			}
			x[ix] = (b[ix] - sum) / aii; //update position of x
		}
		printf("Jacobi iteration %d - x[0] is: %1.9f\r", i, x[0]); //ADD THIS BACK IN
	}
	
	return x;
}

double* jacobi_serial_v2(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	double sum = 0.0;
	double aii = 0.0;
	int indx = 0, next = 0;

	//hipsparseHandle_t sparse_handle;

	//double *x = new double[x_size];
	//double *x0 = new double[x_size];
	//double *xdiff = new double[x_size];
	//double sum = 0.0;
	//double aii = 0.0;
	//int indx = 0, next = 0;

	//for norm function
	hipblasHandle_t blas_handle;
	hipsparseHandle_t sparse_handle;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&sparse_handle);
	hipblasCreate(&blas_handle);
	hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//allocate memory of local variables
	bool *end_flag = (bool*)calloc(1, sizeof(bool));
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	//double* sum = (double*)calloc(1, sizeof(double));
	//double* aii = (double*)calloc(1, sizeof(double));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//declare pointers to device memory
	bool *d_flag;
	long long *d_lrow, *d_lcol;
	double *d_val, *d_x0, *d_b, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_flag, sizeof(double));
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_lrow, sizeof(long long) * n);
	hipMalloc((void **)&d_lcol, sizeof(long long) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	//hipMalloc((void **)&d_sum, sizeof(double));
	//hipMalloc((void **)&d_aii, sizeof(double));
	hipMalloc((void **)&d_b, sizeof(double) * x_size);
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lcol, lcol, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lrow, lrow, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, (sizeof(double) * x_size), hipMemcpyHostToDevice);
	//hipMemcpy(d_x, x, (sizeof(double) * n), hipMemcpyHostToDevice);
	//hipMemcpy(d_xdiff, x_diff, (sizeof(double) * n), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	printf("Size of X: %d\n", x_size);
	int threads = determine_threads(x_size, x_size);
	//int threads = x_size; //what do you think Jacob?
	printf("threads: %d\n", threads);

	//convert input row & col vectors into integer vectors
	init_coo << <1, n >> > (d_lrow, d_lcol, d_row, d_col, n);


	//this is just for testing
	//hipMemcpy(row, d_row, (sizeof(int) * n), hipMemcpyDeviceToHost);
	hipMemcpy(col, d_col, (sizeof(int) * n), hipMemcpyDeviceToHost); //serial only!

	hipDeviceSynchronize();


	//int *csrRowPtr = (int*)calloc(x_size + 1, sizeof(int));
	int *d_rowPtr = 0;

	//for (int i = 0; i < (x_size + 1); i++) {
	//	printf(":%d %d\n", row[i],i);
	//}

	hipMalloc((void**)&d_rowPtr, (x_size + 1) * sizeof(int));
	hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, d_rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(row, d_rowPtr, (sizeof(int) * (x_size + 1)), hipMemcpyDeviceToHost);

	//row = convert_csr(sparse_handle, row, n, x_size);
	hipFree(d_lrow);
	hipFree(d_lcol);
	hipFree(d_rowPtr);

	//for (int i = 0; i < (x_size + 1); i++) {
	//	printf(":%d col:%d at %d\n", row[i], col[i], i);
	//}
	for (int i = 0; i < 2; i++) {
		*end_flag = false;
		//printf("x at iter %d is %f : %f\n", i, *x, *x0);

		std::swap(x0, x);
		

		//THIS IS KERNEL CALL CODE///////////////////////////////////////////////////
		for (int ix = 0; ix < x_size; ix++) {
			
			indx = row[ix];
			next = row[ix + 1];
			
			//printf("at x %d is %d\n", ix, col[indx]);
			sum = 0.0;
			while (indx < next) {
				printf("%d:%d      ", ix, indx);
				if (col[indx] == ix) { //condition is never met
					aii = val[indx];
					//printf("here sum shuld be 0 %d\n", sum);
				}
				else
				{
					sum += ((val[indx]) * x0[col[indx]]); //sum non-zero values in row vec
					//printf("AT x %d, sum is %d\n", ix, sum);
				}
				indx++;
			}

			x[ix] = (b[ix] - sum) / aii;
			x_diff[ix] = x[ix] - x0[ix];
			//if (ix < 10) {
			//	printf("row: %d, x is: %f, sum is: %d\n", ix, x[ix], sum);
			//}
		} //end for (vals of x)
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////
	  //wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);

	return x;
}