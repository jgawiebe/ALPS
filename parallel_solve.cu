#include <stdlib.h>  
#include <stdio.h>
#include <cstdlib>  
#include <hip/hip_runtime.h>

#define BLOCKS 512
#define THREADS 1024

__global__ void kernel_solve(double *val, double *row_ix, double *col_ix, double* b, double* x, unsigned int n) {
	__shared__ int g_val, g_row, g_col, g_x;
	const unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		g_val = val[index];
		g_row = row_ix[index];
		g_col = col_ix[index];
		//result
		x[index] = g_row * g_col;

		printf("Thread %d - value: %d at (%d, %d)\n", index, g_val, g_row, g_col);
	}


}
double* parallel_sor(double *val_ix, double *row_ix, double *col_ix, double *b, const int n) {
	//allocate output vector x
	double *x = new double[n];

	//declare memory size for vectors
	const unsigned int memsize = sizeof(double) * n;

	//declare pointers to device memory
	double *d_val, *d_row, *d_col, *d_b, *d_x;

	//allocate device memory for each vector
	hipMalloc((void **)&d_val, memsize);
	hipMalloc((void **)&d_row, memsize);
	hipMalloc((void **)&d_col, memsize);
	hipMalloc((void **)&d_b, memsize);
	hipMalloc((void **)&d_x, memsize);

	//copy memory and link pointers
	hipMemcpy(d_val, val_ix, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_row, row_ix, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_col, col_ix, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, memsize, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(n / threadsPerBlock.x, n / threadsPerBlock.y);
	
	//execute solve in parallel
	kernel_solve <<<16, 16>>> (d_val, d_row, d_col, d_b, d_x, n);

	//return result
	hipMemcpy(x, d_x, memsize, hipMemcpyDeviceToHost);

	//wait for all processes to complete
	hipDeviceSynchronize();

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_b);
	hipFree(d_x);

	return x;
}

double* serial_sor(double *val, double *row, double *col, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	//allocate output vectors x and x0
	double *x = new double[x_size];
	double *x0 = new double[x_size];
	double sum = 0.0, aii = 0.0;


	for (int i = 0; i < max_iter; i++) {
		x0 = x;

		for (int ix = 0; ix < x_size; ix++) {
			sum = 0; //reset sum
			
			for (int ir = 0; ir < n; ir++) {
				if (row[ir] == ix) {
					if (col[ir] == ix) {
						aii = val[ir];
					}
					else {
						sum += ( val[ir] * x0[(int) col[ir]] ); //sum non-zero values in row vec
					}
				}
			}
			x[ix] = (b[ix] - sum) / aii; //update position of x
		}
	}
	return x;
}