#include "hip/hip_runtime.h"
//This file contains all parallel solving variations using CUDA.
#include <stdlib.h>  
#include <stdio.h>
#include <algorithm> 
#include <cstdlib>  
#include <hip/hip_runtime.h>

//used for norm function
#include <hipblas.h>

//used for conversion from coo to csr sparse matrix formats
#include <hipsparse.h>

#define BLOCKS 512
#define THREADS 1024
#define THREAD_INC 32

//This kernel runs on 1 thread in series, only used for testing CUDA API.
__global__ void kernel_solve_v0(double *val, int *row, int *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	*sum = 0.0;
	
	for (int ir = 0; ir < n; ir++) {
		
		if (row[ir] == ix) {
			if (col[ir] == ix) {
				*aii = val[ir];
			}
			else {
				*sum += (val[ir] * x0[col[ir]]); //sum non-zero values in row vec
			}
		}
	}
}

//Row wise-jacobi solving method (inefficient).
__global__ void kernel_solve_v1(double *val, int *row, int *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	extern __shared__ double thread_sum[]; //access dynamic shared mem

	//initialize thread_sum;
	for (int ir = threadIdx.x; ir < n; ir = ir + blockDim.x) { thread_sum[ir] = 0; }

	__syncthreads();

	for (int ir = threadIdx.x; ir < n; ir = ir + blockDim.x) {
		if (row[ir] == ix) {

			if (col[ir] == ix) {
				*aii = val[ir];
			}
			else
			{
				thread_sum[ir] = ((val[ir]) * x0[col[ir]]); //sum non-zero values in row vec
			}
		}
	}

	__syncthreads();

	//sum all threads and save them to the sum pointer
	if (threadIdx.x == 0) {
		double total = 0.0;
		for (int i = 0; i < n; i++) {
			total += thread_sum[i];
		}
		*sum = total; //return result
	}
}

//Column-wise Jacobi method. NOT COMPLETE. Function does not reach convergence in testing.
__global__ void kernel_solve_v2(double *val, int *row, int *col, double* b, double* x0, double* x, double* xdiff, unsigned int n, unsigned int m, hipblasHandle_t blas_handle, bool *end_flag) {
	double sum = 0.0;
	double aii = 0.0;
	int indx = 0, next = 0;

	//for (int ix = threadIdx.x; ix < m; ix = ix + blockDim.x) {
	for (int ix = 0; ix < m; ix++) {

			indx = row[ix];
			next = row[ix + 1];
			//printf("%d:%d      ", ix, indx);
			//printf("at x %d is %d\n", ix, col[indx]);
			sum = 0.0;
			while (indx < next) {
				if (col[indx] == ix) { //condition is never met
					aii = val[indx];
					//printf("here sum shuld be 0 %d\n", sum);
				}
				else
				{
					sum += ((val[indx]) * x0[col[indx]]); //sum non-zero values in row vec
														  //printf("AT x %d, sum is %d\n", ix, sum);
				}
				indx++;
			}

			x[ix] = (b[ix] - sum) / aii;
			xdiff[ix] = x[ix] - x0[ix];
			printf("Jacobi iteration %d - error is: %f\r", ix, xdiff[ix]); //ADD THIS BACK IN
		} //end for (vals of x)

	//CHECK CONVERGENCE BLOCK NOT DEBUGGED
	//if (threadIdx.x == 0) {
	//	
	//	//compute the norm of vec x
		//hipblasDnrm2(blas_handle, m, x, m, x_norm);
	//	//compute the norm of vec x_diff
	//	hipblasDnrm2(blas_handle, m, xdiff, m, x_diffnorm);

	//	if (*x_diffnorm / *x_norm < 1e-8) {
	//		*end_flag = true;
	//		printf("CONVERGENCE");
	//		return;
	//	}
	//}
	
	
}

//This kernel converts from long long (Armadillo format) to ints for row and column index values.
__global__ void init_coo(long long *lrow, long long *lcol, int* row, int* col, const unsigned int n) {
	for (int i = threadIdx.x; i < n; i = i + blockDim.x) {

		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
	}
}

//Determine optimal number of threads to run based on image size.
int determine_threads(int n, int m) {
	int threads = n / THREAD_INC;
	if ((threads*THREAD_INC) + THREAD_INC <= 1024) {
		threads = (threads*THREAD_INC) + THREAD_INC;
	}
	else {
		threads = 1024;
	}

	if (n * sizeof(double) > 48 * 1024) {
		printf("IMAGE TOO LARGE > EXITING");
		exit(EXIT_FAILURE);
	}

	return threads;
}

//Compute error between iterations using norm function.
double compute_error(double* x, double* x_diff, double* d_x, double* d_xdiff, int x_size, hipblasHandle_t blas_handle) {
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	hipMemcpy(d_x, x, (sizeof(double) * x_size), hipMemcpyHostToDevice);
	hipMemcpy(d_xdiff, x_diff, (sizeof(double) * x_size), hipMemcpyHostToDevice);

	//compute the norm of vec x
	hipblasDnrm2(blas_handle, x_size, d_x, x_size, x_norm);
	//compute the norm of vec x_diff
	hipblasDnrm2(blas_handle, x_size, d_xdiff, x_size, x_diffnorm);

	return *x_diffnorm / *x_norm;
}

//Setup and execution of row-wise Jacobi method.
double* parallel_solve_v2(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter, double tol) {
	//for norm function
	hipblasHandle_t blas_handle;
	hipsparseHandle_t sparse_handle;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&sparse_handle);
	hipblasCreate(&blas_handle);
	hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//allocate memory of local variables
	bool *end_flag = (bool*)calloc(1, sizeof(bool));
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//declare pointers to device memory
	bool *d_flag;
	long long *d_lrow, *d_lcol;
	double *d_val, *d_x0, *d_b, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_flag, sizeof(double));
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_lrow, sizeof(long long) * n);
	hipMalloc((void **)&d_lcol, sizeof(long long) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	hipMalloc((void **)&d_b, sizeof(double) * x_size);
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lcol, lcol, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lrow, lrow, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, (sizeof(double) * x_size), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	int threads = determine_threads(x_size, x_size);
	printf("Solving using parallel Jacobi method\nCurrent matrix contains %d coefficients, solving for %d values\n%d threads assigned\n", n, x_size, threads);

	//convert input row & col vectors into integer vectors
	init_coo << <1, n >> > (d_lrow, d_lcol, d_row, d_col, n);

	hipDeviceSynchronize();

	int *d_rowPtr = 0;

	//convert from coordinate to CSR matrix format
	hipMalloc((void**)&d_rowPtr, (x_size + 1) * sizeof(int));
	hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, d_rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(row, d_rowPtr, (sizeof(int) * (x_size + 1)), hipMemcpyDeviceToHost);

	hipFree(d_lrow);
	hipFree(d_lcol);
	hipFree(d_rowPtr);

	//loop through iterations of Jacobi method
	for (int i = 0; i < max_iter; i++) {
		*end_flag = false;
		
		std::swap(x0, x);
		hipMemcpy(d_x0, x0, (sizeof(double) * x_size), hipMemcpyHostToDevice);
		
		//row wise estimation of x values
		kernel_solve_v2 << <1, threads>> > (d_val, d_row, d_col, d_b, d_x0, d_x, d_xdiff, n, x_size, blas_handle, end_flag);

		hipMemcpy(x, d_x, sizeof(double) * x_size, hipMemcpyDeviceToHost);
		hipMemcpy(end_flag, d_flag, sizeof(bool), hipMemcpyDeviceToHost);

		if (*end_flag) { break; };
	} //end for (iterations)
	printf("\n");

	//wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);

	return x;
}


//Setup and execution of column-wise Jacobi method.
double* parallel_solve_v1(double *val, long long *lrow, long long *lcol, double *b, int *fail, const unsigned int n, const unsigned int x_size, const unsigned int max_iter, double tol) {
	//for norm function
	hipblasHandle_t blas_handle;
	hipblasCreate(&blas_handle);

	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//allocate memory of local variables
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	double* sum = (double*)calloc(1, sizeof(double));
	double* aii = (double*)calloc(1, sizeof(double));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));

	//declare pointers to device memory
	double *d_val, *d_x0, *d_sum, *d_aii, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	hipMalloc((void **)&d_sum, sizeof(double));
	hipMalloc((void **)&d_aii, sizeof(double));
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);

	//convert from long long to int (serially)
	for (int i = 0; i < n; i++) {
		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
	}

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_col, col, (sizeof(int) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_row, row, (sizeof(int) * n), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	int threads = determine_threads(n, x_size);
	printf("Solving using parallel Jacobi method\nCurrent matrix contains %d coefficients, solving for %d values\n%d threads assigned\n", n, x_size, threads);

	//loop for column-wise Jacobi method
	for (int i = 0; i < max_iter; i++) {

		std::swap(x0, x);
		hipMemcpy(d_x0, x0, (sizeof(double) * x_size), hipMemcpyHostToDevice);

		for (int ix = 0; ix < x_size; ix++) {
			*sum = 0.0;

				//execute solve in parallel and allocate n doubles of shared memory
				kernel_solve_v1 << <1, threads, n * sizeof(double) >> > (d_val, d_row, d_col, d_x0, d_sum, d_aii, ix, n);

				//copy shared memory to host (OUTPUTS)
				hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(aii, d_aii, sizeof(double), hipMemcpyDeviceToHost);
				
			x[ix] = (b[ix] - *sum) / *aii;
			x_diff[ix] = x[ix] - x0[ix];
			if (ix < 10) {
				//printf("row: %d, x is: %f:%f, sum is: %f\n", ix, x[ix], x0[ix], *sum);
			}
		} //end for (vals of x)

		//calculate error between x and x_diff
		double error = compute_error(x, x_diff, d_x, d_xdiff, x_size, blas_handle);
		printf("Jacobi iteration %d - error is: %f\r", i, error); //ADD THIS BACK IN

		if (error <= tol) {
			*fail = 0;
			printf("Convergence reached at iter %d, val %f\n", i, *x);
			break; //convergence reached
		}
	} //end for (iterations)

	//wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);
	hipFree(d_sum);
	hipFree(d_aii);

	return x;
}

//Serial execution of Jacobi method (REFRENCE FOR PARALLEL IMPLEMENTATIONS)
double* jacobi_serial(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	//allocate output vectors x and x0
	double *x = new double[x_size];
	double *x0 = new double[x_size];
	double sum = 0.0, aii = 0.0;

	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));

	for (int i = 0; i < n; i++) {
		row[i] = (int)lrow[i];
		col[i] = (int)lcol[i];
		//printf("%d\n", row[i]);
	}

	for (int i = 0; i < max_iter; i++) {
		x0 = x;
		//printf("x at iter %d is %f\n", i, *x);

		for (int ix = 0; ix < x_size; ix++) {
			sum = 0; //reset sum
			
			for (int ir = 0; ir < n; ir++) {
				if (row[ir] == ix) {
					if (col[ir] == ix) {
						aii = val[ir];
					}
					else {
						sum += (val[ir] * x0[col[ir]]); //sum non-zero values in row vec
					}
				}
			}
			x[ix] = (b[ix] - sum) / aii; //update position of x
		}
		printf("Jacobi iteration %d - x[0] is: %1.9f\r", i, x[0]); //ADD THIS BACK IN
	}
	
	return x;
}

//Serial execution of modified Jacobi method suitable for row-wise iterative solving.
double* jacobi_serial_v2(double *val, long long *lrow, long long *lcol, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	double sum = 0.0;
	double aii = 0.0;
	int indx = 0, next = 0;

	//for norm function
	hipblasHandle_t blas_handle;
	hipsparseHandle_t sparse_handle;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&sparse_handle);
	hipblasCreate(&blas_handle);
	hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//allocate memory of local variables
	bool *end_flag = (bool*)calloc(1, sizeof(bool));
	int *row = (int*)calloc(n, sizeof(int));
	int *col = (int*)calloc(n, sizeof(int));
	double* x = (double*)calloc(x_size, sizeof(double));
	double* x0 = (double*)calloc(x_size, sizeof(double));
	double* x_diff = (double*)calloc(x_size, sizeof(double));
	double* x_norm = (double*)calloc(1, sizeof(double));
	double* x_diffnorm = (double*)calloc(1, sizeof(double));

	//declare pointers to device memory
	bool *d_flag;
	long long *d_lrow, *d_lcol;
	double *d_val, *d_x0, *d_b, *d_x, *d_xdiff;
	int *d_row, *d_col;

	//allocate device memory
	hipMalloc((void **)&d_flag, sizeof(double));
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, sizeof(int) * n);
	hipMalloc((void **)&d_col, sizeof(int) * n);
	hipMalloc((void **)&d_lrow, sizeof(long long) * n);
	hipMalloc((void **)&d_lcol, sizeof(long long) * n);
	hipMalloc((void **)&d_x0, sizeof(double) * x_size);
	hipMalloc((void **)&d_b, sizeof(double) * x_size);
	hipMalloc((void **)&d_x, sizeof(double) * x_size);
	hipMalloc((void **)&d_xdiff, sizeof(double) * x_size);

	//copy sparse matrix onto device (INPUTS)
	hipMemcpy(d_val, val, (sizeof(double) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lcol, lcol, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_lrow, lrow, (sizeof(long long) * n), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, (sizeof(double) * x_size), hipMemcpyHostToDevice);

	//determine optimal number of threads while preventing overflow
	printf("Size of X: %d\n", x_size);
	int threads = determine_threads(x_size, x_size);
	printf("threads: %d\n", threads);

	//convert input row & col vectors into integer vectors
	init_coo << <1, n >> > (d_lrow, d_lcol, d_row, d_col, n);

	hipDeviceSynchronize();

	int *d_rowPtr = 0;

	//convert from coordinate to CSR matrix format
	hipMalloc((void**)&d_rowPtr, (x_size + 1) * sizeof(int));
	hipsparseXcoo2csr(sparse_handle, d_row, n, x_size, d_rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	hipMemcpy(row, d_rowPtr, (sizeof(int) * (x_size + 1)), hipMemcpyDeviceToHost);
	
	hipFree(d_lrow);
	hipFree(d_lcol);
	hipFree(d_rowPtr);

	//for (int i = 0; i < (x_size + 1); i++) {
	//	printf(":%d col:%d at %d\n", row[i], col[i], i);
	//}
	for (int i = 0; i < 2; i++) {
		*end_flag = false;
		//printf("x at iter %d is %f : %f\n", i, *x, *x0);

		std::swap(x0, x);
		
		//THIS CODE IS IN THE KERNEL V2
		for (int ix = 0; ix < x_size; ix++) {
			indx = row[ix];
			next = row[ix + 1];

			sum = 0.0;
			while (indx < next) {
				printf("%d:%d      ", ix, indx);
				if (col[indx] == ix) { //condition is never met
					aii = val[indx];
				}
				else
				{
					sum += ((val[indx]) * x0[col[indx]]); //sum non-zero values in row vec
				}
				indx++;
			}
			x[ix] = (b[ix] - sum) / aii;
			x_diff[ix] = x[ix] - x0[ix];
			
		} //end for (vals of x)
	}
 
	//wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(blas_handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_x);
	hipFree(d_xdiff);

	return x;
}
