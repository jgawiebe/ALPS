#include "hip/hip_runtime.h"
#include <stdlib.h>  
#include <stdio.h>
#include <cstdlib>  
#include <hip/hip_runtime.h>

//used for norm function
#include <hipblas.h>
#include <hipblas.h>

#define BLOCKS 512
#define THREADS 1024


__global__ void kernel_solve1(double *val, float *row, float *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	*sum = 0;
	for (int ir = 0; ir < n; ir++) {
		if ((int)row[ir] == ix) {
			if ((int)col[ir] == ix) {
				*aii = val[ir];
			}
			else {
				*sum += (val[ir] * x0[(int)col[ir]]); //sum non-zero values in row vec
			}
		}
	}
}


__global__ void kernel_solve(double *val, float *row, float *col, double* x0, double* sum, double* aii, unsigned int ix, unsigned int n) {
	extern __shared__ double thread_sum[]; //access dynamic shared mem
	//thread_sum = new double[n]; //dynamically allocated array of sums

	//index for each element in the row vector
	//unsigned int row_idx = threadIdx.x;


	for (int row_idx = threadIdx.x; row_idx < n; row_idx = row_idx + blockDim.x)
	{
		thread_sum[row_idx] = 0;
	}
	__syncthreads();


	for (int row_idx = threadIdx.x; row_idx < n; row_idx = row_idx + blockDim.x) {
		if ((int)row[row_idx] == ix) {

			if ((int)col[row_idx] == ix) {
				*aii = val[row_idx];
			}
			else
			{
				thread_sum[row_idx] = ((val[row_idx]) * x0[(int)col[row_idx]]); //sum non-zero values in row vec
				//printf("this is row %0.f at thread %d: val at index is %f\n", row[row_idx], row_idx, val[row_idx]);
			}
		}
	}

	__syncthreads();

	//sum all threads and save them to the sum pointer
	if (threadIdx.x == 0) {
		double total = 0.0;
		for (int i = 0; i < n; i++) {
			total += thread_sum[i];
		}
		*sum = total; //return result
	}
}

double* parallel_sor(double *val, float *row, float *col, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter, double tol) {
	//for norm function
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//declare memory size for vectors
	const unsigned int val_mem = sizeof(double) * n;
	const unsigned int vec_mem = sizeof(float) * n;
	const unsigned int x_mem = sizeof(double) * x_size;
	const unsigned int single = sizeof(double);

	//declare and allocate solver outputs
	double* sum = (double*)malloc(single);
	double* aii = (double*)malloc(single);
	double* x = (double*)malloc(x_mem);
	double* x0 = (double*)calloc(x_size, sizeof(double));
	

	//move these to where they're needed??
	double* x_norm = (double*)malloc(single);
	double* x_diff = (double*)malloc(x_mem);

	//declare pointers to device memory
	double *d_val, *d_x0, *d_sum, *d_aii;
	float *d_row, *d_col;

	//allocate device memory for each vector
	hipMalloc((void **)&d_val, sizeof(double) * n);
	hipMalloc((void **)&d_row, vec_mem);
	hipMalloc((void **)&d_col, vec_mem);
	hipMalloc((void **)&d_x0, x_mem);
	hipMalloc((void **)&d_sum, single);
	hipMalloc((void **)&d_aii, single);

	//copy memory and link pointers (INPUTS)
	hipMemcpy(d_val, val, val_mem, hipMemcpyHostToDevice);
	hipMemcpy(d_row, row, vec_mem, hipMemcpyHostToDevice);
	hipMemcpy(d_col, col, vec_mem, hipMemcpyHostToDevice);
	hipMemcpy(d_x0, x0, x_mem, hipMemcpyHostToDevice);

	//are these needed?
	hipMemcpy(d_aii, aii, single, hipMemcpyHostToDevice);
	hipMemcpy(d_sum, aii, single, hipMemcpyHostToDevice);


	/* Here is full program that is set for parallel running. It always produces sum = 0. There may be an issue with copying 
	mem from device in a loop. We may need to re-copy back to device for each iteration. currently testing with full-fat but
	I need a testing strategy to break it up and check the individual components.*/

	/*At the 3rd iteration, (x=2) the graphics crashes and the outputs start being wrong (I think the actual kernel stops
	executing and the loop continues printing and updating x[ix]*/

	/*first 2 iterations of x are correct. From there x goes off some threshold. This is a memory problem. I need some way to
	free up x0 memory when I'm done each iteration because its eating up all the space.*/

	/*Results vary from totally correct to compeletely random with every run. I have no idea what is causing this.*/
	for (int i = 0; i < 200; i++) {
		x0 = x;
		printf("x at iter %d is %f\n", i, *x);

		if (i > 0) {
			//freeing mem doesn't seem to make a difference
			//hipFree(d_x0);
			//hipMalloc((void **)&d_x0, x_mem);

			hipMemcpy(d_x0, x0, x_mem, hipMemcpyHostToDevice);
		}
		
		for (int ix = 0; ix < x_size; ix++) {
			//*sum = 0.0;

			//execute solve in parallel and allocate n doubles of shared memory
			//check shared mem size n*sizeof(double) <= 48*1024
			//kernel_solve << <1, 128, n*sizeof(double) >> > (d_val, d_row, d_col, d_x0, d_sum, d_aii, ix, n);
			int option = 0;
			if (option == 0) {
				kernel_solve1 << <1, 1 >> > (d_val, d_row, d_col, d_x0, d_sum, d_aii, ix, n);
				//copy shared memory to host (OUTPUTS)
				hipMemcpy(sum, d_sum, single, hipMemcpyDeviceToHost);
				hipMemcpy(aii, d_aii, single, hipMemcpyDeviceToHost);
			}
			else
			{
				*sum = 0; //reset sum

				for (int ir = 0; ir < n; ir++) {
					if ((int)row[ir] == ix) {
						if ((int)col[ir] == ix) {
							*aii = val[ir];
						}
						else {
							*sum += (val[ir] * x0[(int)col[ir]]); //sum non-zero values in row vec
						}
					}
				}
			}

			x[ix] = (b[ix] - *sum) / *aii;
			//x_diff[ix] = x[ix] - x0[ix];
			//printf("x at %d is %f. sum is %0.1f, aii is %0.1f\n", ix, x[ix], *sum, *aii);

		} //end for (vals of x)

		//compute the norm of vec x
		hipblasDnrm2(handle, x_size, x, 0, x_norm);
		//compute the norm of x_diff
		hipblasDnrm2(handle, x_size, x_diff, 0, x_diff); //put norm in first location of x_diff

		//check if error is within tolerance
		double error = *x_diff / *x_norm;
		if (error <= tol) {
			printf("convergence...");
			break; //convergence reached
		} //end if

	} //end for (iterations)

	//wait for all processes to complete
	hipDeviceSynchronize();

	hipblasDestroy(handle);

	//free device memory
	hipFree(d_val);
	hipFree(d_row);
	hipFree(d_col);
	hipFree(d_x0);
	hipFree(d_sum);
	hipFree(d_aii);

	return x;
}

double* serial_sor(double *val, float *row, float *col, double *b, const unsigned int n, const unsigned int x_size, const unsigned int max_iter) {
	//allocate output vectors x and x0
	double *x = new double[x_size];
	double *x0 = new double[x_size];
	double sum = 0.0, aii = 0.0;


	for (int i = 0; i < 50; i++) {
		x0 = x;
		printf("x at iter %d is %f\n", i, *x);

		for (int ix = 0; ix < x_size; ix++) {
			sum = 0; //reset sum
			
			for (int ir = 0; ir < n; ir++) {
				if (row[ir] == ix) {
					if (col[ir] == ix) {
						aii = val[ir];
					}
					else {
						sum += (val[ir] * x0[(int)col[ir]]); //sum non-zero values in row vec
					}
				}
			}
			x[ix] = (b[ix] - sum) / aii; //update position of x
		}
	}
	return x;
}